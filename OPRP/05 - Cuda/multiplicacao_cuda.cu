#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "toolsv3.h"

#define T 1024 // numero max de threads por bloco

__global__ void matMult (int *da, int *db, int *dc, int *C_dev, int *Cb_dev, int *La_dev) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int j = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(i<*La_dev && j<*Cb_dev){
	  int soma=0;
      for(int ii=0; ii< *C_dev ; ii++){
        soma += da[i*(*C_dev)+ii] * db[ii*(*Cb_dev)+j];
      }
    dc[i*(*Cb_dev)+j] = soma;
  }
}

__global__ void printIndex (void) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = blockIdx.y * blockDim.y + threadIdx.y;
   printf ("[%d][%d]\t(x)\t%d\t%d\t%d\t(y)\t%d\t%d\t%d\n",i,j, threadIdx.x, blockIdx.x, blockDim.x,threadIdx.y, blockIdx.y, blockDim.y);
}

__global__ void dirtyMem (int *da) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   da[i] = 0;
}

__host__ void initvet(int *host_a, mymatriz mat_a) {
  int linha_a =mat_a.lin, coluna_a=mat_a.col;
  for (int i=0; i < linha_a; i++) {
    for (int j=0; j < coluna_a; j++) {
     host_a[i*coluna_a+j] = mat_a.matriz[i][j];
    }
  }
}

__host__ void printMat (int *mat, int lin, int col){
	for (int j=0; j < lin && j<15; j++)
	printf("\t(%d)", j);
	printf("\n\n");
	for (int i=0; i < lin && i<15; i++) {
		printf("(%d)", i);
		for (int j=0; j < col && j<15; j++){
			printf("\t%d", mat[i*col+j]);
		}
		printf("\n\n\n");
	}
}

__host__ mymatriz *mmultiplicar (mymatriz *mat_a, mymatriz *mat_b) {
	mymatriz *mat_c = NULL;
	if (mat_a->col != mat_b->lin){
		printf ("Erro: Matrizes são incompatíveis!\n");
		exit(1);
	}
	mat_c = (mymatriz *) malloc (sizeof(mymatriz));
	mat_c->lin = mat_a->lin;
	mat_c->col = mat_b->col;
	if (malocar(mat_c)) {	printf ("ERROR: Out of memory\n"); }
		//a(linhas) x b(colunas)
		printf("Multiplicando com ijk\n" );
		for (int i=0; i <mat_a->lin; i++){
			for (int j=0; j <mat_b->col; j++){
				mat_c->matriz[i][j]=0;
				for (int k=0; k < mat_b->lin; k++){
					mat_c->matriz[i][j] += mat_a->matriz[i][k]*mat_b->matriz[k][j];
				}
			}
		}
    return mat_c;
}

__host__ int mcomparar (mymatriz *mat_a, int *vet_c, int col){
	for (int j =0; j < mat_a->col; j++)
	for (int i=0; i < mat_a->lin; i++) {
		for (int j=0; j < mat_a->col; j++){
			if (mat_a->matriz[i][j] != vet_c[i*col+j]) {
				printf("O elemento [%d,%d] é diferente nas matrizes analisadas!", i,j);
				return 1;
			}
		}
	}
	printf("\tVERIFICADO: Matrizes são idênticas\n");
	return 0;
}

__host__ int mimprimir (mymatriz *matriz){
	int linha, coluna;
	linha = matriz->lin;
	coluna = matriz->col;
	if (linha > 15) {
		linha = 15;
	}
	if (coluna > 15) {
		coluna = 15;
	}
	for (int j =0; j < coluna; j++)
	printf("\t(%d)", j);
	printf("\n");
	for (int i=0; i < linha; i++) {
		printf("(%d)", i);
		for (int j=0; j < coluna; j++){
			printf("\t%d", matriz->matriz[i][j]);
		}
		printf("\n");
	}
	printf("\n \
	matriz->lin-1, matriz->col-1, matriz->matriz[matriz->lin-1][matriz->col-1]);
	return 0;
}

int main(int argc, char const *argv[]) {
  mymatriz mat_a, mat_b;
  FILE *fmat;
  int nr_line;
  int *vet_line = NULL;
  int L, C, Ca, Lb;

  if (argc != 3){
		printf ("ERRO: Numero de parametros %s <matriz_a> <matriz_b>\n", argv[0]);
		exit (1);
	}

  fmat = fopen(argv[1],"r");

  if (fmat == NULL) {
      printf("Error: Na abertura dos arquivos.");
      exit(1);
  }

  extrai_parametros_matriz(fmat, &L, &Ca, &vet_line, &nr_line);
  mat_a.matriz = NULL;
  mat_a.lin = L;
  mat_a.col = Ca;
  if (malocar(&mat_a)) {
    printf ("ERROR: Out of memory\n");
  }
  filein_matriz (mat_a.matriz, L, Ca, fmat, vet_line, nr_line);
  free (vet_line);
  fclose(fmat);

  fmat = fopen(argv[2],"r");
  if (fmat == NULL) {
    printf("Error: Na abertura dos arquivos.");
    exit(1);
  }

  extrai_parametros_matriz(fmat, &Lb, &C, &vet_line, &nr_line);
  mat_b.matriz = NULL;
  mat_b.lin = Lb;
  mat_b.col = C;
  if (malocar(&mat_b)) {
      printf ("ERROR: Ficamos sem memória galera ;-;\n");
  }
  filein_matriz (mat_b.matriz, Lb, C, fmat, vet_line, nr_line);

  free (vet_line);
  fclose(fmat);

  if(Ca != Lb){
    printf("Error: Matrizes são incompativeis\n");
    exit(1);
  }

  printf("Multiplicacao sequencial\n");
  double tempo_s=wtime();
  mymatriz *mult_sequencial = mmultiplicar(&mat_a, &mat_b);
  tempo_s = wtime()-tempo_s;

  int *a, *b, *c;
  int *devA, *devB, *devC;
  int tamanhoA, tamanhoB, tamanhoC;

  tamanhoA = L * Ca * sizeof(int);
  tamanhoB = Lb * C * sizeof(int);
  tamanhoC = L * C * sizeof(int);

  hipError_t error_c;
  error_c = hipHostMalloc((void **) &a, tamanhoA);

  if(error_c != hipSuccess){ printf("GPUassert: %s\n", hipGetErrorString(error_c)); }
  error_c=hipHostMalloc((void **) &b, tamanhoB);
  if(error_c != hipSuccess){ printf("GPUassert: %s\n", hipGetErrorString(error_c));}
  error_c=hipHostMalloc((void **) &c, tamanhoC);
  if(error_c != hipSuccess){ printf("GPUassert: %s\n", hipGetErrorString(error_c));}

  initvet(a, mat_a);
  initvet(b, mat_b);

  printf ("\t ### Valores Lidos de arquivo na CPU ###\n");
  printf ("\t ### Matriz (a) ### \n");
  printMat(a, L, Ca);
  printf ("\t ### Matriz (b) ### \n");
  printMat(b,Lb, C);

  hipMalloc ((void **) &devA, tamanhoA);
  hipMalloc ((void **) &devB, tamanhoB);
  hipMalloc ((void **) &devC, tamanhoC);

  dim3 dimBlock (1, 1);
  dim3 dimThreads(L, Ca);

  if(L*Ca > T){
	dimThreads.x=(int) ceil(sqrt(T));
    dimThreads.y=(int) ceil(sqrt(T));
    dimBlock.x= (int) ceil(double(L)/sqrt(T));//sqrt(1024)
    dimBlock.y= (int) ceil(double(Ca)/sqrt(T));//sqrt(1024) pois 32*32=1024 e precisamos L*C < 1024
  }

  dirtyMem<<<dimBlock, dimThreads>>>(devA);

	dimThreads.x=Lb;
  dimThreads.y=C;
  dimBlock.x= 1;
  dimBlock.y= 1;

  if(Lb*C > T){
	  dimThreads.x=(int) ceil(sqrt(T));
    dimThreads.y=(int) ceil(sqrt(T));
    dimBlock.x= (int) ceil(double(Lb)/sqrt(T));//sqrt(1024)
    dimBlock.y= (int) ceil(double(C)/sqrt(T));//sqrt(1024) pois 32*32=1024 e precisamos L*C < 1024
  }

dirtyMem<<<dimBlock, dimThreads>>>(devB);

dimThreads.x=L;
dimThreads.y=C;
dimBlock.x= 1;
dimBlock.y= 1;

if(L*C > T){
	dimThreads.x=(int) ceil(sqrt(T));
  dimThreads.y=(int) ceil(sqrt(T));
  dimBlock.x= (int) ceil(double(L)/sqrt(T));
  dimBlock.y= (int) ceil(double(C)/sqrt(T));
}
dirtyMem<<<dimBlock, dimThreads>>>(devC);

hipMemcpy (devA, a, tamanhoA, hipMemcpyHostToDevice);
hipMemcpy (devB, b, tamanhoB, hipMemcpyHostToDevice);

dimThreads.x=L;
dimThreads.y=C;
dimBlock.x= 1;
dimBlock.y= 1;

  if(L*C > T){
	dimThreads.x=(int) ceil(sqrt(T));
	dimThreads.y=(int) ceil(sqrt(T));
	dimBlock.x= (int) ceil(double(L)/sqrt(T));
	dimBlock.y= (int) ceil(double(C)/sqrt(T));
  }

  int *C_dev, *Cb_dev, *La_dev;//c_dev = Ca (matrizA) e Cb_dev=C (matrizb), La_dev = L (matrixA)
  hipMalloc((void **) &C_dev, sizeof(int));
  hipMalloc((void **) &Cb_dev, sizeof(int));
  hipMalloc((void **) &La_dev, sizeof(int));
  hipMemcpy (C_dev, &Ca, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Cb_dev, &C, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(La_dev, &L, sizeof(int), hipMemcpyHostToDevice);

  printf("Mult CUDA\n");
  double tempo_c = wtime();
  matMult<<< dimBlock, dimThreads>>>(devA, devB, devC, C_dev, Cb_dev, La_dev);

  hipDeviceSynchronize();

  hipMemcpy (c, devC, tamanhoC, hipMemcpyDeviceToHost);
  tempo_c = wtime()- tempo_c;

  printf ("\t ### [CUDA] Matriz (c) ### \n");
  printMat(c, L, C);
  printf("\n\t ### [sequencial] Matriz ###\n");
  mimprimir(mult_sequencial);

  printf("Comparando as matrizes:\n");
  mcomparar(mult_sequencial, c, C);
  printf("\nBlocos(%d,%d) threads(%d,%d)\n", dimBlock.x, dimBlock.y, dimThreads.x, dimThreads.y);
  printf("Tempo de execucao sequencial: %.3f\n", tempo_s);
  printf("Tempo de execucao CUDA: %.3f\n", tempo_c);
  printf("SpeedUp sequencial/GPU: %.3f\n", tempo_s/tempo_c);

  hipFree(devA);
  hipFree(devB);
  hipFree(devC);
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);

  return 0;
}
